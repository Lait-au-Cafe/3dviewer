#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "kernel.h"

__global__ void devStoreVertices(
	float* Vertex,
	const int width,
	const int num_vertex
) {
	const int tx = blockIdx.x*blockDim.x + threadIdx.x;
	const int ty = blockIdx.y*blockDim.y + threadIdx.y;
	const int tz = blockIdx.z*blockDim.z + threadIdx.z;
	
	uint coord = tx + ty * width;
	if(tx >= width || coord >= num_vertex){
		return;
	}

	float vx, vy, vz;
	vx = (float)tx * 0.1 - 0.5;
	vy = (float)ty * 0.1 - 0.5;
	vz = (float)tz * 0.05;

	vx = 0.95 * vx + 0.31 * vz;
	vz = -0.31 * vx + 0.95 * vz;

	coord = tx + ty * width + tz * num_vertex;
	uint index;
	index = 3 * coord;
	Vertex[index] = vx;
	index = 3 * coord + 1;
	Vertex[index] = vy;
	index = 3 * coord + 2;
	Vertex[index] = vz;
}

void StoreVertices(
	float* vertex,
	const int width,
	const int num_vertex,
	const int layers
){
	const int height = (num_vertex - 1) / width + 1;

	// define thread / block size
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(
			(width - 1) / dimBlock.x + 1, 
			(height - 1) / dimBlock.y + 1, 
			layers);

	std::cout
		<< "\n== Configs of StoreVertex ==\n"
		<< "Num of Vertices : " << num_vertex << "\n"
		<< "Width : " << width << "\n"
		<< "Height : " << height << "\n"
		<< "Dim of Grid : (" 
			<< dimGrid.x << ", " << dimGrid.y << ", " << dimGrid.z << ")\n"
		<< "Dim of Block : (" 
			<< dimBlock.x << ", " << dimBlock.y << ", " << dimBlock.z << ")\n"
		<< std::endl;

	devStoreVertices<<<dimGrid, dimBlock, 0 >>>(vertex, width, num_vertex);
	return;
}

__global__ void devMLS(
	float* Vertex,
	const int width,
	const int height,
	const int layers,
	const int window,
	const float radius
) {
	const int tx = blockIdx.x*blockDim.x + threadIdx.x;
	const int ty = blockIdx.y*blockDim.y + threadIdx.y;
	const int tz = blockIdx.z*blockDim.z + threadIdx.z;

	int min_x = max(0, tx - window);
	int max_x = min(width - 1, tx + window);
	int min_y = max(0, ty - window);
	int max_y = min(height - 1, ty + window);
	int min_z = max(0, tz - window);
	int max_z = min(layers - 1, tz + window);

	uint coord, index;
	float vx, vy, vz;
	for(int z = min_z; z <= max_z; z++){
	for(int y = min_y; y <= max_y; y++){
	for(int x = min_x; x <= max_x; x++){
		coord = x + y * width + z * width * height;

		index = 3 * coord;
		vx = Vertex[index];
		index = 3 * coord + 1;
		vy = Vertex[index];
		index = 3 * coord + 2;
		vz = Vertex[index];
	}}}

}

void MLS(
	float* vertex,
	const int width,
	const int height,
	const int layers,
	const int window,
	const float radius
) {
	// define thread / block size
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(
			(width - 1) / dimBlock.x + 1, 
			(height - 1) / dimBlock.y + 1, 
			(layers - 1) / dimBlock.z + 1);

	devMLS<<<dimGrid, dimBlock, 0>>>(vertex, width, height, layers, window, radius);
}
